
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

extern "C" __global__ void hello_world(void) {
    printf("Hello World! from thread [%d,%d] on GPGPU\n", threadIdx.x, blockIdx.x);
}
